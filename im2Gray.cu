#include "hip/hip_runtime.h"
#include "im2Gray.h"

#define BLOCK 16



/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols){

 /*
   Your kernel here: Make sure to check for boundary conditions
  */

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  

  if (x < numCols && y < numRows){
    int grayOffset = y * numCols + x;
    unsigned char r = d_in[grayOffset].x; 
    unsigned char g = d_in[grayOffset].y;
    unsigned char b = d_in[grayOffset].z;
    d_grey[grayOffset] = 0.299f * (float)r + 0.587f * (float)g + 0.114f * (float)b;
  } 
}




void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols){
    // configure launch params here 
    
    dim3 block(BLOCK, BLOCK, 1);
    dim3 grid((numCols-1)/BLOCK + 1, (numRows-1)/BLOCK + 1, 1);
    
    im2Gray<<<grid,block>>>(d_in, d_grey, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    
}





