#include "hip/hip_runtime.h"
#include "im2Gray.h"

#define BLOCK 256



/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols){

 /*
   Your kernel here: Make sure to check for boundary conditions
  */

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if (x < numCols && y < numRows){
    int grayOffset= y*numCols + x;
    int rgbOffset= grayOffset*3;
    //unsigned char r = d_in[rgbOffset]; 
    //unsigned char g = d_in[rgbOffset + 1];
    //unsigned char b = d_in[rgbOffset + 2];
    char r = d_in[rgbOffset]; 
    char g = d_in[rgbOffset + 1];
    char b = d_in[rgbOffset + 2];
    d_grey[grayOffset] = 0.299f*r + 0.587f*g + 0.114f*b;
  } 
}




void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols){
    // configure launch params here 
    
    dim3 block(BLOCK,1,1);
    dim3 grid((numRows*numCols + 1)/BLOCK,1,1);

    im2Gray<<<grid,block>>>(d_in, d_grey, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    
}





