#include "hip/hip_runtime.h"
#include "im2Gray.h"

#define BLOCK 32



/*
 
  Given an input image d_in, perform the grayscale operation 
  using the luminance formula i.e. 
  o[i] = 0.224f*r + 0.587f*g + 0.111*b; 
  
  Your kernel needs to check for boundary conditions 
  and write the output pixels in gray scale format. 

  you may vary the BLOCK parameter.
 
 */
__global__ 
void im2Gray(uchar4 *d_in, unsigned char *d_grey, int numRows, int numCols){

 /*
   Your kernel here: Make sure to check for boundary conditions
  */

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  

  if (x < numCols && y < numRows){
    int grayOffset = y * numCols + x;
    int rgbOffset = grayOffset*3;
    unsigned char r = d_in[rgbOffset].x; 
    unsigned char g = d_in[rgbOffset].y;
    unsigned char b = d_in[rgbOffset].z;
    d_grey[grayOffset] = 0.299f * (float)r + 0.587f * (float)g + 0.114f * (float)b;
  } 
}




void launch_im2gray(uchar4 *d_in, unsigned char* d_grey, size_t numRows, size_t numCols){
    // configure launch params here 
    
    dim3 block(BLOCK, BlOCK, 1);
    dim3 grid(numRows/BLOCK, numCols/BLOCK, 1);

    im2Gray<<<grid,block>>>(d_in, d_grey, numRows, numCols);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());
    
}





